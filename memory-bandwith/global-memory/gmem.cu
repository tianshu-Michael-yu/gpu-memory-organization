
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "assert.h"

typedef unsigned short int u16;
typedef unsigned int u32;

#define CUDA_CALL(x) {const hipError_t a = (x); if (a != hipSuccess) { printf("\nCUDA Error: %s (err_num=%d)\n", hipGetErrorString(a), a); hipDeviceReset(); assert(0);}}

#define KERNEL_LOOP 4096

__device__ static u32 gmem_data_gpu[KERNEL_LOOP];
static u32 const_data_host[KERNEL_LOOP];

__global__ void const_test_gpu_gmem(u32 * const data, const u32 num_elements) {
    const u32 tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < num_elements) {
        u32 d = gmem_data_gpu[0];

        for (int i=0; i<KERNEL_LOOP; i++) {
            d ^= gmem_data_gpu[i];
            d |= gmem_data_gpu[i];
            d &= gmem_data_gpu[i];
            d |= gmem_data_gpu[i];
        }

        data[tid] = d;
    }
}

void cuda_error_check() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        assert(0);
    }
}

__host__ void generate_rand_data(u32 * host_data_ptr) {
    for (int i=0; i<KERNEL_LOOP; i++) {
        host_data_ptr[i] = rand();
    }
}


int main() {
    const u32 num_elements = 1 << 20;
    const u32 threads_per_block = 256;
    const u32 num_blocks = (num_elements + threads_per_block - 1) / threads_per_block;


    generate_rand_data(const_data_host);

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gmem_data_gpu), const_data_host, sizeof(u32) * KERNEL_LOOP));
    u32 * data_gpu;
    CUDA_CALL(hipMalloc(&data_gpu, sizeof(*data_gpu) * num_elements));

    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreateWithFlags(&stop, hipEventBlockingSync));

    CUDA_CALL(hipEventRecord(start));
    const_test_gpu_gmem<<<num_blocks, threads_per_block>>>(data_gpu, num_elements);
    cuda_error_check();

    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));
    float elapsed_time;
    CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, stop));

    printf("gloabl memory time: %f ms\n", elapsed_time);

    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));

    CUDA_CALL(hipFree(data_gpu));

    return EXIT_SUCCESS;
}