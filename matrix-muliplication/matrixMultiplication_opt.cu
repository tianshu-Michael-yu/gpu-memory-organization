#include "hip/hip_runtime.h"
#include "matrixMultiplication.h"
#include <stdio.h>

__global__ void matrixMulKernel(int *d_A, int *d_B, int *d_C, size_t size) {
    int i= blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size && k < size) {
        for (int j=0; j<size; j++) {
            atomicAdd(&d_C[i*size + j], d_A[i*size + k] * d_B[k*size + j]);
        }
    }
}


void matrixMultiplication(int *matrixA, int *matrixB, int *matrixC, size_t matrixSize) {
    int *d_A, *d_B, *d_C;

    // Allocate memory on the device
    hipError_t err1 = hipMalloc((void **) &d_A, matrixSize * matrixSize * sizeof(*d_A));
    hipError_t err2 = hipMalloc((void **) &d_B, matrixSize * matrixSize * sizeof(*d_B));
    hipError_t err3 = hipMalloc((void **) &d_C, matrixSize * matrixSize * sizeof(*d_C));

    // error checking
    if (err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess) {
        printf("Error allocating memory on the device\n");
        exit(EXIT_FAILURE);
    }

    // Copy the matrices to the device
    hipError_t err4 = hipMemcpy(d_A, matrixA, matrixSize * matrixSize * sizeof(*d_A), hipMemcpyHostToDevice);
    hipError_t err5 = hipMemcpy(d_B, matrixB, matrixSize * matrixSize * sizeof(*d_B), hipMemcpyHostToDevice);



    // error checking
    if (err4 != hipSuccess || err5 != hipSuccess) {
        printf("Error copying memory to the device\n");
        exit(EXIT_FAILURE);
    }
    
    // Launch the kernel
    dim3 dimBlock(32, 32);
    dim3 dimGrid((matrixSize + dimBlock.x - 1) / dimBlock.x, (matrixSize + dimBlock.y - 1) / dimBlock.y);
    matrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, matrixSize);

    // Copy the result back to the host
    hipError_t err6 = hipMemcpy(matrixC, d_C, matrixSize * matrixSize * sizeof(*d_C), hipMemcpyDeviceToHost);

    // error checking
    if (err6 != hipSuccess) {
        printf("Error copying memory back to the host\n");
        exit(EXIT_FAILURE);
    }

    // Free device memory
    hipError_t err7 = hipFree(d_A);
    hipError_t err8 = hipFree(d_B);
    hipError_t err9 = hipFree(d_C);

    // error checking
    if (err7 != hipSuccess || err8 != hipSuccess || err9 != hipSuccess) {
        printf("Error freeing memory on the device\n");
        exit(EXIT_FAILURE);
    }
}