#include "hip/hip_runtime.h"
#include "vectorAdd.h"
#include <time.h>
#include <stdio.h>
#define TPB 1024

__global__ void vectorAddKernel(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void vectorAdd(int *a, int *b, int *c, int n) {
    int *dev_a, *dev_b, *dev_c;

    // allocate memory on device
    hipMalloc((void**)&dev_a, n * sizeof(int));
    hipMalloc((void**)&dev_b, n * sizeof(int));
    hipMalloc((void**)&dev_c, n * sizeof(int));

    // copy vectors from host memory to device memory
    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    clock_t kernel_start = clock();
    // launch kernel
    vectorAddKernel<<<ceil(n / (float)TPB), TPB>>>(dev_a, dev_b, dev_c, n);
    hipDeviceSynchronize();
    clock_t kernel_end = clock();
    printf("Kernel time: %f\n", (double)(kernel_end - kernel_start) / CLOCKS_PER_SEC);

    // copy result from device memory to host memory
    hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}